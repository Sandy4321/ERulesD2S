#include "hip/hip_runtime.h"
#include "parameters.h"
#include "functions.cu"

SEMAPHORE wait_sem[MAX_THREADS],post_sem[MAX_THREADS];
CUTThread threadID[MAX_THREADS];
Plan plan[MAX_THREADS];

int numClasses, numberAttributes, currentnumberInstances, maxnumberInstances, maxnumberInstances_A, numThreads, populationSize, classifiedClass;
bool evaluate = false;
bool copy = false;
float *h_instancesData;
jobject algorithm;

void releaseGPU(JNIEnv *env, jobject obj)
{
	evaluate = false;
	copy = false;
	
    // Wake up threads to finish them
	for(int i = 0; i < numThreads; i++)
		SEM_POST (&wait_sem[i]);
		
	free(h_instancesData);
	
	cutWaitForThreads(threadID, numThreads);

	#if _WIN32
	for(int i = 0; i < numThreads; i++)
	{
		CloseHandle(wait_sem[i]);
		CloseHandle(post_sem[i]);
	}
	#endif
}

static void Get_VM(JavaVM** jvm_p, JNIEnv** env_p) {

	JavaVM jvmBuffer;
	JavaVM* vmBuf = &jvmBuffer;
	jsize jvmTotalNumberFound = 0;  
	jint resCheckVM = JNI_GetCreatedJavaVMs(&vmBuf, 1, &jvmTotalNumberFound);
	
	if (jvmTotalNumberFound < 1)
	{
		fprintf(stderr, "No JVM found\n");
		exit(0);
	}
	*jvm_p = vmBuf;

	(*jvm_p)->AttachCurrentThread((void**)env_p, NULL);
}

__global__ void coverageKernel(unsigned char* result, float* instancesData, int* rulesConsequent, int currentnumberInstances, int maxnumberInstances_A, int numberAttributes, float* expressions) 
{
   int instance = blockDim.y * blockIdx.y + threadIdx.y;
   int resultMemPosition = blockIdx.x * maxnumberInstances_A + instance;
   
   if(instance < currentnumberInstances)
   {
      if(covers(&expressions[MAX_EXPR_LEN * blockIdx.x], instance, instancesData, maxnumberInstances_A))
      {
         if(rulesConsequent[blockIdx.x] == instancesData[(numberAttributes-1)*maxnumberInstances_A + instance])
            result[resultMemPosition] = 0; // TRUE POSITIVE
         else
            result[resultMemPosition] = 2; // FALSE POSITIVE
      }
      else
      {
         if(rulesConsequent[blockIdx.x] != instancesData[(numberAttributes-1)*maxnumberInstances_A + instance])
            result[resultMemPosition] = 1; // TRUE NEGATIVE
         else
            result[resultMemPosition] = 3; // FALSE NEGATIVE   
      }
   }
}

__global__ void fitnessKernel(unsigned char* result, int currentnumberInstances, int maxnumberInstances_A, float* fitness) 
{
   __shared__ int MC[512];
   
   MC[threadIdx.y] = 0;
   MC[threadIdx.y+128] = 0;
   MC[threadIdx.y+256] = 0;
   MC[threadIdx.y+384] = 0;
   
   int base = blockIdx.x*maxnumberInstances_A + threadIdx.y;
   int top =  blockIdx.x*maxnumberInstances_A + currentnumberInstances - base;
   
   // Performs the reduction of the thread corresponding values
   for(int i = 0; i < top; i+=128)
   {
      MC[threadIdx.y*4 + result[base + i]]++;
   }
   
   __syncthreads();
   
    // Calculates the final amount
   if(threadIdx.y < 4)
   {
      for(int i = 4; i < 512; i+=4)
      {
         MC[0] += MC[i];     // Number of true positives
         MC[1] += MC[i+1];   // Number of true negatives
         MC[2] += MC[i+2];   // Number of false positives
         MC[3] += MC[i+3];   // Number of false negatives
      }
   }
   
   if(threadIdx.y == 0)
   {
      int tp = MC[0], tn = MC[1], fp = MC[2], fn = MC[3];
      
      float se, sp;

	  if(tp + fn == 0)
		se = 1.0f;
	  else
		se = tp / (float) (tp + fn);

	  if(tn + fp == 0)
		sp = 1.0f;
	  else
		sp = tn / (float) (tn + fp);

      // Set the fitness to the individual
      fitness[blockIdx.x] = se * sp;
   }
}

CUT_THREADPROC gpuThreadTan(Plan *plan)
{
	hipError_t err;
	
	hipSetDevice(plan->device);
	
	int threadPopulationSize;
	
	float *d_rules, *h_rules;
	float *d_instancesData;
	int *d_rulesConsequent;
	int *h_rulesConsequent;
	
	float *h_fitness, *d_fitness;
	
	unsigned char* d_result;
	
	JNIEnv* env;
	JavaVM* jvm;
	
	// Signal: thread is ready to evaluate
	SEM_POST(&post_sem[plan->thread]);
	
	hipMalloc((void**) &d_rules, BLOCK_SIZE_RULES * MAX_EXPR_LEN * sizeof(float));
	hipMalloc((void**) &d_instancesData, numberAttributes * maxnumberInstances_A * sizeof(float));
	hipMalloc((void**) &d_rulesConsequent, BLOCK_SIZE_RULES * sizeof(int));
	hipMalloc((void**) &d_fitness, BLOCK_SIZE_RULES * sizeof(float));
    hipMalloc((void**) &d_result, BLOCK_SIZE_RULES * maxnumberInstances_A * sizeof(unsigned char));
    
	hipHostMalloc((void**)&h_rules, BLOCK_SIZE_RULES *  MAX_EXPR_LEN * sizeof(float));
	hipHostMalloc((void**)&h_rulesConsequent, BLOCK_SIZE_RULES *  sizeof(int));
	hipHostMalloc((void**)&h_fitness, BLOCK_SIZE_RULES * sizeof(float));
	
	err = hipGetLastError();
	
    if(hipSuccess != err)
    {
    	printf( "Cuda error: %s.\n",  hipGetErrorString( err) );
    	exit(0);
    }
    
	Get_VM(&jvm, &env);
	
	dim3 threads_coverage(1, THREADS_EVAL_BLOCK);
	dim3 threads_fitness(1, 128);
	
	do
	{
		// Wait until evaluation is required
		SEM_WAIT (&wait_sem[plan->thread]);
		
		if(evaluate)
		{
			// Get the methods from Java
			jclass cls = env->GetObjectClass(algorithm);
			jmethodID getAntecedent = env->GetMethodID(cls, "getAntecedent", "(I)[F");
			jmethodID setFitness = env->GetMethodID(cls, "setFitness", "(IF)V");
			jmethodID getConsequent = env->GetMethodID(cls, "getConsequent", "(I)I");
		
			// Calculate the thread population size
			threadPopulationSize = (int)ceil(populationSize/(float)numThreads);
			
			// If population overflow, recalculate the thread actual population size
			if((plan->thread + 1) * threadPopulationSize > populationSize)
			{
				if((threadPopulationSize = populationSize - threadPopulationSize * plan->thread) < 0)
					threadPopulationSize = 0;
			}
			if(threadPopulationSize > 0)
			{
				// Calculate the base index of the individual for this thread
				int base = plan->thread * (int)ceil(populationSize/(float)numThreads);
				
				int numberIndstoEvaluate = BLOCK_SIZE_RULES;	
				
				// Population is evaluated using blocks of BLOCK_SIZE_RULES individuals
				for(int j = 0; j < threadPopulationSize; j += BLOCK_SIZE_RULES)
				{
					// If the last block size is smaller, fix the block size to the number of the rest of individuals 
					if(j+BLOCK_SIZE_RULES > threadPopulationSize)
						numberIndstoEvaluate = threadPopulationSize - j;
					
					memset(h_rules, 0, BLOCK_SIZE_RULES * MAX_EXPR_LEN * sizeof(float));
					
					for(int i = 0; i < numberIndstoEvaluate; i++)
					{
						jfloatArray antecedent = (jfloatArray) env->CallObjectMethod(algorithm, getAntecedent, base+j+i); 
  						
  						float *antecedentElements = (float*) env->GetFloatArrayElements(antecedent, 0);
  						
					    memcpy(&h_rules[i*MAX_EXPR_LEN], &antecedentElements[1], (antecedentElements[0] * sizeof(float)));
					    
					    env->ReleaseFloatArrayElements(antecedent, antecedentElements, 0);
					    env->DeleteLocalRef(antecedent);
					    
						h_rulesConsequent[i] = env->CallIntMethod(algorithm, getConsequent, base+j+i);
					}
					
					hipMemcpy(d_rules, h_rules, BLOCK_SIZE_RULES * MAX_EXPR_LEN * sizeof(float), hipMemcpyHostToDevice);
					hipMemcpy(d_rulesConsequent , h_rulesConsequent, BLOCK_SIZE_RULES * sizeof(int), hipMemcpyHostToDevice);
					
					// Setup evaluation grid size	
					dim3 grid_coverage(numberIndstoEvaluate, (int)ceil(currentnumberInstances/(float)THREADS_EVAL_BLOCK));
					dim3 grid_fitness(numberIndstoEvaluate, 1);
					
					coverageKernel <<< grid_coverage, threads_coverage >>> (d_result, d_instancesData, d_rulesConsequent, currentnumberInstances, maxnumberInstances_A, numberAttributes, d_rules);
					
					fitnessKernel <<< grid_fitness, threads_fitness >>> (d_result, currentnumberInstances, maxnumberInstances_A, d_fitness);
					
	                hipMemcpy(h_fitness, d_fitness, numberIndstoEvaluate * sizeof(float), hipMemcpyDeviceToHost );
                
	                for(int i = 0; i < numberIndstoEvaluate; i++)
	               		env->CallVoidMethod(algorithm, setFitness, base + j + i, h_fitness[i]);
				}
			}
		}
		else if(copy)
		{
		    // Copy instances data and classes to the GPU
			hipMemcpy(d_instancesData, h_instancesData, numberAttributes*maxnumberInstances_A*sizeof(float), hipMemcpyHostToDevice );
		}
		else
		{
			 // Algorithm finished, free dynamic memory
			 hipFree(d_rules);
	         hipFree(d_instancesData);
	         hipFree(d_rulesConsequent);
	         hipFree(d_fitness);
    		 hipFree(d_result);
	         
	         hipHostFree(h_rules);
	         hipHostFree(h_rulesConsequent);
	         hipHostFree(h_fitness);
		}
		
		// Evaluation finished
		SEM_POST(&post_sem[plan->thread]);
		
	}while(evaluate || copy);

	jvm->DetachCurrentThread();
	
	CUT_THREADEND;
}

JNIEXPORT void JNICALL
Java_net_sf_jclec_problem_classification_evolutionarylearner_RuleEvaluatorGPU_releaseGPU(JNIEnv *env, jobject obj)
{
	releaseGPU(env,obj);
}

JNIEXPORT void JNICALL
Java_net_sf_jclec_problem_classification_evolutionarylearner_RuleEvaluatorGPU_allocateMemoryGPU(JNIEnv *env, jobject obj, jint popSize, jint jmaxnumberInstances, jint jnumberAttributes, jint jnumClasses, jobject jalgorithm)
{
	algorithm = jalgorithm;
	numberAttributes = jnumberAttributes;
	numClasses = jnumClasses;
	maxnumberInstances = jmaxnumberInstances;
	maxnumberInstances_A = ceil(maxnumberInstances/(float)ALIGNMENT)*ALIGNMENT;
	
	h_instancesData = (float*)malloc(numberAttributes*maxnumberInstances_A*sizeof(float));

	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int deviceCountComputeCapability = 0;
	int deviceID[16];
	
	for (int dev = 0; dev < deviceCount; dev++)
    {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);
        
        //if((deviceProp.major == 3 && deviceProp.minor == 5) || (deviceProp.major == 5 && deviceProp.minor == 0) || (deviceProp.major == 5 && deviceProp.minor == 2) || (deviceProp.major == 6 && deviceProp.minor == 0) || (deviceProp.major == 6 && deviceProp.minor == 1))
        {
       		deviceID[deviceCountComputeCapability] = dev;
        	deviceCountComputeCapability++;
        }
    }
    
    if(deviceCountComputeCapability < 1)
	{
		fprintf(stderr, "CUDA devices count is %d\n", deviceCount);
		exit(0);
	}
    
    numThreads = deviceCountComputeCapability;
	
	// Set up semaphores
	for(int i = 0; i < numThreads; i++)
	{
		SEM_INIT (&wait_sem[i], 0);
		SEM_INIT (&post_sem[i], 0);

		plan[i].thread = i;
		plan[i].device = deviceID[i];
		plan[i].size = (int)ceil(popSize/(float)numThreads);
	}
	
	for(int i = 0; i < numThreads; i++)
		threadID[i] = cutStartThread((CUT_THREADROUTINE) gpuThreadTan, (void *)&plan[i]);

	// SIGNAL: threads ready to evaluate
	for(int i = 0; i < numThreads; i++)
		SEM_WAIT (&post_sem[i]);
}

JNIEXPORT void JNICALL
Java_net_sf_jclec_problem_classification_evolutionarylearner_RuleEvaluatorGPU_copyDatasetGPU(JNIEnv *env, jobject obj, jint jcurrentnumberInstances, jobject jalgorithm)
{
	algorithm = jalgorithm;
	currentnumberInstances = jcurrentnumberInstances;
	jclass cls = env->GetObjectClass(algorithm);
	jmethodID getValue = env->GetMethodID(cls, "getValue", "(II)F");
	
	// Copy dataset data from Java
	for(int i = 0; i < jcurrentnumberInstances; i++)
		for(int j = 0; j < numberAttributes; j++)
			h_instancesData[j*maxnumberInstances_A+i] = env->CallFloatMethod(algorithm,getValue,i,j);
			
	copy = true;
	evaluate = false;
	
	// SIGNAL: wake up threads to copy
	for(int i = 0; i < numThreads; i++)
       SEM_POST (&wait_sem[i]);
	
	// Wait until threads finish
	for(int i = 0; i < numThreads; i++)
	   SEM_WAIT (&post_sem[i]);
}

JNIEXPORT void JNICALL
Java_net_sf_jclec_problem_classification_evolutionarylearner_RuleEvaluatorGPU_evaluateGPU(JNIEnv *env, jobject obj, jint size, jobject jalgorithm)
{
	algorithm = jalgorithm;
	populationSize = size;
	
	copy = false;
	evaluate = true;

	// SIGNAL: wake up threads to evaluate
	for(int i = 0; i < numThreads && i < size; i++)
       SEM_POST (&wait_sem[i]);
	
	// Wait until threads finish
	for(int i = 0; i < numThreads && i < size; i++)
	   SEM_WAIT (&post_sem[i]);
}
